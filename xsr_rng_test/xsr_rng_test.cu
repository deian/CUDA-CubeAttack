#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdio.h>
#include <stdint.h>
#include "../xsr_rng.h"

typedef uint32_t u32;

__global__ static void test_inline_xsr_rand_u32(u32 *seed, int N, u32 *out) {
   int i;
   int tid=blockIdx.x*blockDim.x+threadIdx.x;
   inline_xsr_def_u32();
   inline_xsr_srand_u32(seed[tid]);
   for(i=0;i<N-1;i++) {
      inline_xsr_rand_u32();
   }
   out[tid]=inline_xsr_rand_u32();
}


int test_inline_xsr(int dev, int nr_threads, int nr_blocks,int N) {
   int i,j;
   u32 *seed_h,*seed_d;
   u32 *out_h,*out_d;
   hipDeviceProp_t deviceProp;
   unsigned int timer = 0;
   double proc_time=-1.0;
   size_t size=(nr_threads*nr_blocks)*sizeof(u32);

   hipGetDeviceProperties(&deviceProp, dev);
   printf("\nUsing device %d: \"%s\"\n", dev, deviceProp.name);
   printf("\nClock rate: %d\n",deviceProp.clockRate);
   hipSetDevice(dev);

   cutilSafeCall(hipHostAlloc((void**)&(seed_h),size,hipHostMallocPortable));
   cutilSafeCall(hipMalloc((void**)&(seed_d),size));

   cutilSafeCall(hipHostAlloc((void**)&(out_h),size,hipHostMallocPortable));
   cutilSafeCall(hipMalloc((void**)&(out_d),size));

   {
      FILE *fp;
      if(!(fp = fopen("/dev/urandom", "r"))) {
         fprintf(stderr,"Failed to open /dev/urandom");
         return -1;
      }
      fread(seed_h,sizeof(u32),nr_threads*nr_blocks,fp);
      fclose(fp);
   }

   cutilCheckError(cutCreateTimer( &timer));
   cutilCheckError(cutStartTimer( timer));
#define NR_TEST_ITERATIONS 1//000
   for(int nr_test_iter=0;nr_test_iter<NR_TEST_ITERATIONS;nr_test_iter++) 
   {


      cutilSafeCall(hipMemcpy(seed_d,seed_h,size,hipMemcpyHostToDevice));

      test_inline_xsr_rand_u32<<<nr_blocks,nr_threads>>>(seed_d,N,out_d);
      cutilCheckMsg("Kernel execution failed");
      hipDeviceSynchronize();

      cutilSafeCall(hipMemcpy(out_h,out_d,size,hipMemcpyDeviceToHost));


   }
   cutilCheckError(cutStopTimer( timer));
   proc_time=cutGetTimerValue( timer);
   cutilCheckError(cutDeleteTimer( timer));

   for(i=0;i<nr_threads*nr_blocks;i++) {
      u32 out_val=0;
      xsr_srand_u32(seed_h[i]);
      for(j=0;j<N;j++) { out_val=xsr_rand_u32(); }
      if(out_val!=out_h[i]) {
         printf("failed @ [%4d] -> seed=%08x cpu=%08x gpu=%08x\n",i,
                                             seed_h[i],out_val,out_h[i]);
      }
   }


   cutilSafeCall(hipFree(seed_d));
   cutilSafeCall(hipFree(out_d));
   cutilSafeCall(hipHostFree(seed_h));
   cutilSafeCall(hipHostFree(out_h));

   printf( "\nProcessing time: %8g (ms), %g (us/byte) %g (cycles/byte)\n"
         ,proc_time
         ,(1000.0*proc_time)/(NR_TEST_ITERATIONS*N)/
                           (nr_threads*nr_blocks*sizeof(u32))
         ,((proc_time)*(deviceProp.clockRate))/(NR_TEST_ITERATIONS*N)/
                           (nr_threads*nr_blocks*sizeof(u32)));


   return 0;
}
int main(void) {
   return test_inline_xsr(3,256,256,100000);
}
