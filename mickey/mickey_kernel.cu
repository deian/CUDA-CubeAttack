#include "hip/hip_runtime.h"
#ifndef __D_MICKEY_KERNEL__
#define __D_MICKEY_KERNEL__
#include "mickey.h"


#define R_Mask0   0x1279327b
#define R_Mask1   0xb5546660
#define R_Mask2   0xdf87818f
#define R_Mask3   0x00000003

#define Comp00    0x6aa97a30
#define Comp01    0x7942a809
#define Comp02    0x057ebfea
#define Comp03    0x00000006
#define Comp10    0xdd629e9a
#define Comp11    0xe3a21d63
#define Comp12    0x91c23dd7
#define Comp13    0x00000001

#define S_Mask00  0x9ffa7faf
#define S_Mask01  0xaf4a9381
#define S_Mask02  0x9cec5802
#define S_Mask03  0x00000001

#define S_Mask10  0x4c8cb877
#define S_Mask11  0x4911b063
#define S_Mask12  0x40fbc52b
#define S_Mask13  0x00000008

#define d_mickey_state(R,S) \
   R##0,R##1,R##2,R##3,S##0,S##1,S##2,S##3

#define state(X,a)      X##a


#define CLOCK_R(R,input_bit,control_bit)                                   \
{                                                                          \
    int Feedback_bit;                                                      \
    int Carry0, Carry1, Carry2;                                            \
                                                                           \
    Feedback_bit = ((state(R,3) >> 3) & 1) ^ (input_bit);                  \
    Carry0 = (state(R,0) >> 31) & 1;                                       \
    Carry1 = (state(R,1) >> 31) & 1;                                       \
    Carry2 = (state(R,2) >> 31) & 1;                                       \
                                                                           \
    if ((control_bit)) {                                                   \
        state(R,0) ^= (state(R,0) << 1);                                   \
        state(R,1) ^= (state(R,1) << 1) ^ Carry0;                          \
        state(R,2) ^= (state(R,2) << 1) ^ Carry1;                          \
        state(R,3) ^= (state(R,3) << 1) ^ Carry2;                          \
    } else {                                                               \
        state(R,0) = (state(R,0) << 1);                                    \
        state(R,1) = (state(R,1) << 1) ^ Carry0;                           \
        state(R,2) = (state(R,2) << 1) ^ Carry1;                           \
        state(R,3) = (state(R,3) << 1) ^ Carry2;                           \
    }                                                                      \
                                                                           \
    if (Feedback_bit) {                                                    \
       state(R,0) ^= R_Mask0;                                              \
       state(R,1) ^= R_Mask1;                                              \
       state(R,2) ^= R_Mask2;                                              \
       state(R,3) ^= R_Mask3;                                              \
    }                                                                      \
}

#define CLOCK_S(S,input_bit,control_bit)                                   \
{                                                                          \
    int Feedback_bit;                                                      \
    int Carry0, Carry1, Carry2;                                            \
                                                                           \
    Feedback_bit = ((state(S,3) >> 3) & 1) ^ (input_bit);                  \
    Carry0 = (state(S,0) >> 31) & 1;                                       \
    Carry1 = (state(S,1) >> 31) & 1;                                       \
    Carry2 = (state(S,2) >> 31) & 1;                                       \
                                                                           \
    state(S,0) = (state(S,0) << 1) ^ ((state(S,0) ^ Comp00) &              \
          ((state(S,0) >> 1) ^ (state(S,1) << 31) ^ Comp10) & 0xfffffffe); \
    state(S,1) = (state(S,1) << 1) ^ ((state(S,1) ^ Comp01) &              \
          ((state(S,1) >> 1) ^ (state(S,2) << 31) ^ Comp11)) ^ Carry0;     \
    state(S,2) = (state(S,2) << 1) ^ ((state(S,2) ^ Comp02) &              \
          ((state(S,2) >> 1) ^ (state(S,3) << 31) ^ Comp12)) ^ Carry1;     \
    state(S,3) = (state(S,3) << 1) ^ ((state(S,3) ^ Comp03) &              \
          ((state(S,3) >> 1) ^ Comp13) & 0x7) ^ Carry2;                    \
                                                                           \
    if (Feedback_bit) {                                                    \
        if ((control_bit)) {                                               \
            state(S,0) ^= S_Mask10;                                        \
            state(S,1) ^= S_Mask11;                                        \
            state(S,2) ^= S_Mask12;                                        \
            state(S,3) ^= S_Mask13;                                        \
        } else {                                                           \
            state(S,0) ^= S_Mask00;                                        \
            state(S,1) ^= S_Mask01;                                        \
            state(S,2) ^= S_Mask02;                                        \
            state(S,3) ^= S_Mask03;                                        \
        }                                                                  \
    }                                                                      \
}

#define CLOCK_KG_INIT(R,S,input_bit)                                       \
{                                                                          \
    int control_bit_r;                                                     \
    int control_bit_s;                                                     \
                                                                           \
    control_bit_r = ((state(S,1) >> 2) ^ (state(R,2) >> 3)) & 1;           \
    control_bit_s = ((state(R,1) >> 1) ^ (state(S,2) >> 3)) & 1;           \
                                                                           \
    CLOCK_R(R, ((state(S,1) >> 18) & 1) ^ (input_bit), control_bit_r);     \
    CLOCK_S(S, (input_bit), control_bit_s);                                \
                                                                           \
}


#define CLOCK_KG_KS(ks,R,S)                                                \
{                                                                          \
    int control_bit_r;                                                     \
    int control_bit_s;                                                     \
                                                                           \
    ks = (state(R,0) ^ state(S,0)) & 1;                                    \
    control_bit_r = ((state(S,1) >> 2) ^ (state(R,2) >> 3)) & 1;           \
    control_bit_s = ((state(R,1) >> 1) ^ (state(S,2) >> 3)) & 1;           \
                                                                           \
    CLOCK_R(R, 0, control_bit_r);                                          \
    CLOCK_S(S, 0, control_bit_s);                                          \
}


#define min(a,b) ((a)<(b))?(a):(b)

__device__ __host__ u32 d_mickey(u32 key0, u32 key1, u32 key2,
                                 u32  iv0, u32  iv1, u32  iv2,
                                 u32 nr_output_bits) {

   int i,z;
   int iv_size=MICKEY_IV_SIZE;
   u32 output=0;



   /* define state */
   u32 d_mickey_state(R,S);

   /* initialize R and S */
   state(R,0)=state(R,1)=state(R,2)=state(R,3)=0;
   state(S,0)=state(S,1)=state(S,2)=state(S,3)=0;

   /* - load IV ----------------------------------------------------- */
   {
      int nr_iv_clks=min(iv_size,32);
#if (MICKEY_IV_SIZE>=32)
#pragma unroll 32
#endif
      for(i=0;i<nr_iv_clks;i++) { CLOCK_KG_INIT(R,S,(iv0>>(31-i))&1); }
      iv_size-=nr_iv_clks; nr_iv_clks=min(iv_size,32);
#if (MICKEY_IV_SIZE>=64)
#pragma unroll 32
#endif
      for(i=0;i<nr_iv_clks;i++) { CLOCK_KG_INIT(R,S,(iv1>>(31-i))&1); }
      iv_size-=nr_iv_clks; nr_iv_clks=min(iv_size,32);
#if (MICKEY_IV_SIZE==80)
#pragma unroll 16
#endif
      for(i=0;i<nr_iv_clks;i++) { CLOCK_KG_INIT(R,S,(iv2>>(31-i))&1); }
   }
   /* --------------------------------------------------------------- */


   /* - load key ---------------------------------------------------- */
   {
#pragma unroll 32 
      for(i=0;i<32;i++) { CLOCK_KG_INIT(R,S,(key0>>(31-i))&1); }
#pragma unroll 32 
      for(i=0;i<32;i++) { CLOCK_KG_INIT(R,S,(key1>>(31-i))&1); }
#pragma unroll 16 
      for(i=0;i<16;i++) { CLOCK_KG_INIT(R,S,(key2>>(31-i))&1); }
   }
   /* --------------------------------------------------------------- */

   /* - preclock ---------------------------------------------------- */
#if ((NR_INIT_ROUNDS%100)==0)
   #pragma unroll 100
#elif ((NR_INIT_ROUNDS%32)==0)
   #pragma unroll 32 
#elif ((NR_INIT_ROUNDS%16)==0)
   #pragma unroll 16
#elif ((NR_INIT_ROUNDS%8)==0)
   #pragma unroll 8
#elif ((NR_INIT_ROUNDS%4)==0)
   #pragma unroll 4
#elif ((NR_INIT_ROUNDS%2)==0)
   #pragma unroll 2
#endif
    for(i=0;i<NR_INIT_ROUNDS;i++) { CLOCK_KG_INIT(R,S,0); }
   /* --------------------------------------------------------------- */

   /* - generate keystream ------------------------------------------ */

#if (NR_TEST_ITERATIONS>1)
#pragma unroll 1
   for(int nr_test_iter=0;nr_test_iter<NR_TEST_ITERATIONS;nr_test_iter++)  {
   output=0;
#endif

#ifdef NR_OUTPUT_BITS
   #if NR_OUTPUT_BITS==32
      #pragma unroll 32 
   #elif NR_OUTPUT_BITS==16
      #pragma unroll 16
   #elif NR_OUTPUT_BITS==8
      #pragma unroll 8
   #elif NR_OUTPUT_BITS==4
      #pragma unroll 4
   #elif NR_OUTPUT_BITS==2
      #pragma unroll 2
   #endif
#endif
   for(i=0;i<nr_output_bits-1;i++) {
      CLOCK_KG_KS(z,R,S)
      output|=(z<<(31-i));
   }
   z=(state(R,0) ^ state(S,0)) & 1;
   output|=(z<<(31-i));
#if (NR_TEST_ITERATIONS>1)
   }
#endif
   /* --------------------------------------------------------------- */

   return output;
}


/*
   For our application we a only care about the word-length on
   the GPU, which is 32-bits, so this implementation is slightly
   limited though it's quite easy to extend.
 */
int d_mickey_test(char *fname) {
   u32 z;
   FILE *fp;

   int i,nr_tests;
   u32 test_key[3],test_iv[3],test_z;

   if(!(fp=fopen(fname,"r"))) {
      fprintf(stderr,"d_mickey_test: Failed to open \'%s\'\n", fname);
      return -1;
   }

   fscanf(fp,"%u\n",&nr_tests);
   debug("nr_tests=%d\n",nr_tests);

   for(i=0;i<nr_tests;i++) {
    fscanf(fp,"%x,%x,%x,%x,%x,%x,%x,\n",&test_key[0],&test_key[1],&test_key[2],
                                         &test_iv[0],&test_iv[1],&test_iv[2],
                                         &test_z);

#define ch_endianess(a)                                 \
{                                                       \
   a=(((a>>24)&0x000000ff) | ((a>> 8)&0x0000ff00)       \
    | ((a<< 8)&0x00ff0000) | ((a<<24)&0xff000000));     \
}
      ch_endianess(test_key[0]);        ch_endianess(test_iv[0]);
      ch_endianess(test_key[1]);        ch_endianess(test_iv[1]);
      ch_endianess(test_key[2]);        ch_endianess(test_iv[2]);
      ch_endianess(test_z);

#undef ch_endianess


      z=d_mickey(test_key[0],test_key[1],test_key[2],
                  test_iv[0],test_iv[1],test_iv[2],NR_OUTPUT_BITS);

      if(z^( test_z & (0xFFFFFFFF>>(32-NR_OUTPUT_BITS)))) {
         fprintf(stderr,"d_mickey_test: Failed test number %2d:"
               "%08x!=%08x\n",i,z,test_z);
         fclose(fp);
         return -1;
      }
   }

   fclose(fp);
   return 0;
}

/*
int main() {
         d_mickey_test("mickey.long.test.32");
   return 0;
}
*/
#define black_box_key(key)                                      \
   key##0,key##1,key##2

#define black_box_iv(iv)                                        \
   iv##0,iv##1,iv##2

#define black_box_clear_key(key)                                \
   key##0=key##1=key##2=0;                                      \

#define black_box_clear_iv(iv)                                  \
   iv##0=iv##1=iv##2=0;                                         \

#define black_box_id2iv(iv,I,nr_idx,threadID)                   \
{                                                               \
  u32 _i;                                                       \
  u64 tid=(threadID);                                           \
  black_box_clear_iv(iv);                                       \
  for(_i=0;_i<(nr_idx);_i++) {                                  \
     u32 mask = (tid&0x1)<<((I)[_i]-32*((I)[_i]/32));           \
     switch(((I)[_i]/32)) {                                     \
      case 0: iv##0 |= mask; break;                             \
      case 1: iv##1 |= mask; break;                             \
      case 2: iv##2 |= mask; break;                             \
     }                                                          \
     tid>>=1;                                                   \
  }                                                             \
}                                                    

#define black_box_key_set_bitpos(key,bitpos)                    \
{                                                               \
   u32 mask=(0x1)<<((bitpos)-32*((bitpos)/32));                 \
   switch(((bitpos)/32)) {                                      \
      case 0: key##0 |= mask; break;                            \
      case 1: key##1 |= mask; break;                            \
      case 2: key##2 |= mask; break;                            \
   }                                                            \
}                                                    

#define black_box_key_random(key,rand)                          \
{                                                               \
   key##0=rand();                                               \
   key##1=rand();                                               \
   key##2=rand();                                               \
}

#define black_box_add_keys(keyD,key0,key1)                      \
{                                                               \
   keyD##0=key0##0^key1##0;                                     \
   keyD##1=key0##1^key1##1;                                     \
   keyD##2=key0##2^key1##2;                                     \
}

/* - general macros ------------------------------------------ */
#define black_box_def_key(key)                                  \
   u32 black_box_key(key);

#define black_box_def_iv(iv)                                    \
   u32 black_box_iv(iv);

#define black_box_def_state(key,iv)                             \
   black_box_def_key(key);                                      \
   black_box_def_iv(iv);                                        \
   black_box_clear_key(key);                                    \
   black_box_clear_iv(iv);                                      

#define black_box_test(fname)                                   \
   d_mickey_test(fname)

#define black_box_query(key,iv,nr_output_bits)                  \
   d_mickey(black_box_key(key),black_box_iv(iv),nr_output_bits)
/* ----------------------------------------------------------- */
#endif
